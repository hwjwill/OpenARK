#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <sstream>
#include <string>
#include <fstream>
#include <unordered_map>

#include "tsdf.cuh"

// CUDA kernel function to integrate a TSDF voxel volume given depth images
namespace ark
{
    __global__
    void Integrate(float * cam_K, float * cam2base, float * depth_im,
                   int im_height, int im_width, int voxel_grid_dim_x, int voxel_grid_dim_y, int voxel_grid_dim_z,
                   float voxel_grid_origin_x, float voxel_grid_origin_y, float voxel_grid_origin_z, float voxel_size, float trunc_margin,
                   float * voxel_grid_TSDF, float * voxel_grid_weight) {

        int pt_grid_z = blockIdx.x;
        int pt_grid_y = threadIdx.x;

        for (int pt_grid_x = 0; pt_grid_x < voxel_grid_dim_x; ++pt_grid_x) {

            // Convert voxel center from grid coordinates to base frame camera coordinates
            float pt_base_x = voxel_grid_origin_x + pt_grid_x * voxel_size;
            float pt_base_y = voxel_grid_origin_y + pt_grid_y * voxel_size;
            float pt_base_z = voxel_grid_origin_z + pt_grid_z * voxel_size;

            // Convert from base frame camera coordinates to current frame camera coordinates
            float tmp_pt[3] = {0};
            tmp_pt[0] = pt_base_x - cam2base[0 * 4 + 3];
            tmp_pt[1] = pt_base_y - cam2base[1 * 4 + 3];
            tmp_pt[2] = pt_base_z - cam2base[2 * 4 + 3];
            float pt_cam_x = cam2base[0 * 4 + 0] * tmp_pt[0] + cam2base[1 * 4 + 0] * tmp_pt[1] + cam2base[2 * 4 + 0] * tmp_pt[2];
            float pt_cam_y = cam2base[0 * 4 + 1] * tmp_pt[0] + cam2base[1 * 4 + 1] * tmp_pt[1] + cam2base[2 * 4 + 1] * tmp_pt[2];
            float pt_cam_z = cam2base[0 * 4 + 2] * tmp_pt[0] + cam2base[1 * 4 + 2] * tmp_pt[1] + cam2base[2 * 4 + 2] * tmp_pt[2];

            if (pt_cam_z <= 0)
                continue;

            int pt_pix_x = roundf(cam_K[0 * 3 + 0] * (pt_cam_x / pt_cam_z) + cam_K[0 * 3 + 2]);
            int pt_pix_y = roundf(cam_K[1 * 3 + 1] * (pt_cam_y / pt_cam_z) + cam_K[1 * 3 + 2]);
            if (pt_pix_x < 0 || pt_pix_x >= im_width || pt_pix_y < 0 || pt_pix_y >= im_height)
                continue;

            float depth_val = depth_im[pt_pix_y * im_width + pt_pix_x];

            if (depth_val <= 0 || depth_val > 6)
                continue;

            float diff = depth_val - pt_cam_z;

            if (diff <= -trunc_margin)
                continue;

            // Integrate
            int volume_idx = pt_grid_z * voxel_grid_dim_y * voxel_grid_dim_x + pt_grid_y * voxel_grid_dim_x + pt_grid_x;
            float dist = fmin(1.0f, diff / trunc_margin);
            float weight_old = voxel_grid_weight[volume_idx];
            float weight_new = weight_old + 1.0f;
            voxel_grid_weight[volume_idx] = weight_new;
            voxel_grid_TSDF[volume_idx] = (voxel_grid_TSDF[volume_idx] * weight_old + dist) / weight_new;
        }
    }
__host__
GpuTsdfGenerator::GpuTsdfGenerator(int width, int height, float fx, float fy, float cx, float cy,
                                   float v_g_o_x = -1.5f, float v_g_o_y = -1.5f, float v_g_o_z = 0.5f,
                                   float v_size = 0.006f, float trunc_m = 0.03f, int v_g_d_x = 500, int v_g_d_y = 500, int v_g_d_z = 500){
    im_width_ = width;
    im_height_ = height;

    memset(p_cam_K_, 0.0f, sizeof(float) * 3*3);
    p_cam_K_[0] = fx;
    p_cam_K_[2] = cx;
    p_cam_K_[4] = fy;
    p_cam_K_[5] = cy;
    p_cam_K_[8] = 1.0f;

    voxel_grid_origin_x_ = v_g_o_x;
    voxel_grid_origin_y_ = v_g_o_y;
    voxel_grid_origin_z_ = v_g_o_z;

    voxel_grid_dim_x_ = v_g_d_x;
    voxel_grid_dim_y_ = v_g_d_y;
    voxel_grid_dim_z_ = v_g_d_z;

    voxel_size_ = v_size;

    trunc_margin_ = trunc_m;

    std::cout << "fx: " <<fx<<std::endl;
    std::cout << "cx: " <<cx<<std::endl;
    std::cout << "fy: " <<fy<<std::endl;
    std::cout << "cy: " <<cy<<std::endl;

    // Initialize voxel grid
    p_voxel_grid_TSDF_ = new float[voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_];
    p_voxel_grid_weight_ = new float[voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_];
    memset(p_voxel_grid_TSDF_, 1.0f, sizeof(float) * voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_);
    memset(p_voxel_grid_weight_, 0.0f, sizeof(float) * voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_);

    // Load variables to GPU memory
    hipMalloc(&p_gpu_voxel_grid_TSDF_, voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_ * sizeof(float));
    hipMalloc(&p_gpu_voxel_grid_weight_, voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_ * sizeof(float));
    checkCUDA(__LINE__, hipGetLastError());
    hipMemcpy(p_gpu_voxel_grid_TSDF_,p_voxel_grid_TSDF_, voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_ * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(p_gpu_voxel_grid_weight_, p_voxel_grid_weight_, voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_ * sizeof(float), hipMemcpyHostToDevice);
    checkCUDA(__LINE__, hipGetLastError());
    hipMalloc(&p_gpu_cam_K_, 3 * 3 * sizeof(float));
    hipMemcpy(p_gpu_cam_K_,p_cam_K_, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&p_gpu_cam2base_, 4 * 4 * sizeof(float));
    hipMalloc(&p_gpu_depth_im_, im_height_ * im_width_ * sizeof(float));
    checkCUDA(__LINE__, hipGetLastError());
}

__host__
void GpuTsdfGenerator::processFrame(float *depth_im, float *cam2base)
{
    hipMemcpy(p_gpu_cam2base_, cam2base, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(p_gpu_depth_im_, depth_im, im_height_ * im_width_ * sizeof(float), hipMemcpyHostToDevice);
    checkCUDA(__LINE__, hipGetLastError());

    Integrate <<< voxel_grid_dim_z_, voxel_grid_dim_y_ >>>(p_gpu_cam_K_, p_gpu_cam2base_, p_gpu_depth_im_,
          im_height_, im_width_, voxel_grid_dim_x_, voxel_grid_dim_y_, voxel_grid_dim_z_,
          voxel_grid_origin_x_, voxel_grid_origin_y_, voxel_grid_origin_z_, voxel_size_, trunc_margin_,
            p_gpu_voxel_grid_TSDF_, p_gpu_voxel_grid_weight_);
    checkCUDA(__LINE__, hipGetLastError());
}

__host__
void GpuTsdfGenerator::Shutdown() {

}

__host__
void GpuTsdfGenerator::hello()
{
    std::cout << "Hello World" <<std::endl;
}

__host__
void GpuTsdfGenerator::SaveTSDF(std::string filename) {
    // Load TSDF voxel grid from GPU to CPU memory
    hipMemcpy(p_voxel_grid_TSDF_, p_gpu_voxel_grid_TSDF_, voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_ * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(p_voxel_grid_weight_, p_gpu_voxel_grid_weight_, voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_ * sizeof(float), hipMemcpyDeviceToHost);
    checkCUDA(__LINE__, hipGetLastError());
    // Save TSDF voxel grid and its parameters to disk as binary file (float array)
    std::cout << "Saving TSDF voxel grid values to disk (tsdf.bin)..." << std::endl;
    std::string voxel_grid_saveto_path = filename;
    std::ofstream outFile(voxel_grid_saveto_path, std::ios::binary | std::ios::out);
    float voxel_grid_dim_xf = (float) voxel_grid_dim_x_;
    float voxel_grid_dim_yf = (float) voxel_grid_dim_y_;
    float voxel_grid_dim_zf = (float) voxel_grid_dim_z_;
    outFile.write((char*)&voxel_grid_dim_xf, sizeof(float));
    outFile.write((char*)&voxel_grid_dim_yf, sizeof(float));
    outFile.write((char*)&voxel_grid_dim_zf, sizeof(float));
    outFile.write((char*)&voxel_grid_origin_x_, sizeof(float));
    outFile.write((char*)&voxel_grid_origin_y_, sizeof(float));
    outFile.write((char*)&voxel_grid_origin_z_, sizeof(float));
    outFile.write((char*)&voxel_size_, sizeof(float));
    outFile.write((char*)&trunc_margin_, sizeof(float));
    for (int i = 0; i < voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_; ++i)
        outFile.write((char*)&p_voxel_grid_TSDF_[i], sizeof(float));
    outFile.close();
}

__host__
void GpuTsdfGenerator::SavePLY(std::string filename) {
    hipMemcpy(p_voxel_grid_TSDF_, p_gpu_voxel_grid_TSDF_, voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_ * sizeof(float), hipMemcpyDeviceToHost);
//    hipMemcpy(p_voxel_grid_weight_, p_gpu_voxel_grid_weight_, voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_ * sizeof(float), hipMemcpyDeviceToHost);

    checkCUDA(__LINE__, hipGetLastError());
    tsdf2mesh(filename);
}

    __host__
    XYZ GpuTsdfGenerator::VertexInterp(float isolevel, XYZ p1, XYZ p2, float valp1, float valp2)
    {
        float mu;
        XYZ p;

        if (fabs(isolevel - valp1) < 0.00001)
            return p1;
        if (fabs(isolevel - valp2) < 0.00001)
            return p2;
        if (fabs(valp1 - valp2) < 0.00001)
            return p1;
        mu = (isolevel - valp1) / (valp2 - valp1);
        p.x = p1.x + mu * (p2.x - p1.x);
        p.y = p1.y + mu * (p2.y - p1.y);
        p.z = p1.z + mu * (p2.z - p1.z);

        return p;
    }

    __host__
    void GpuTsdfGenerator::tsdf2mesh(std::string outputFileName) {
        int totalSize = voxel_grid_dim_x_ * voxel_grid_dim_y_ * voxel_grid_dim_z_;
        std::vector<FACE> faces;
        std::vector<XYZ> vertices;
        std::unordered_map<std::string, int> verticesIdx;
        int vertexCount = 0;
        for (size_t i = 0; i < totalSize; ++i) {
            int xi = i / (voxel_grid_dim_y_ * voxel_grid_dim_z_);
            int yi = (i - xi * voxel_grid_dim_y_ * voxel_grid_dim_z_) / voxel_grid_dim_z_;
            int zi = i - xi * voxel_grid_dim_y_ * voxel_grid_dim_z_ - yi * voxel_grid_dim_z_;
            if (xi == voxel_grid_dim_x_ - 1 || yi == voxel_grid_dim_y_ - 1 || zi == voxel_grid_dim_z_ - 1)
                continue;
            GRIDCELL grid;
            grid.p[0] = XYZ(xi, yi, zi);
            grid.p[1] = XYZ(xi, yi + 1, zi);
            grid.p[2] = XYZ(xi + 1, yi + 1, zi);
            grid.p[3] = XYZ(xi + 1, yi, zi);
            grid.p[4] = XYZ(xi, yi, zi + 1);
            grid.p[5] = XYZ(xi, yi + 1, zi + 1);
            grid.p[6] = XYZ(xi + 1, yi + 1, zi + 1);
            grid.p[7] = XYZ(xi + 1, yi, zi + 1);

            grid.val[0] = p_voxel_grid_TSDF_[xi * voxel_grid_dim_y_ * voxel_grid_dim_z_ + yi * voxel_grid_dim_z_ + zi];
            grid.val[1] = p_voxel_grid_TSDF_[xi * voxel_grid_dim_y_ * voxel_grid_dim_z_ + (yi + 1) * voxel_grid_dim_z_ + zi];
            grid.val[2] = p_voxel_grid_TSDF_[(xi + 1) * voxel_grid_dim_y_ * voxel_grid_dim_z_ + (yi + 1) * voxel_grid_dim_z_ + zi];
            grid.val[3] = p_voxel_grid_TSDF_[(xi + 1) * voxel_grid_dim_y_ * voxel_grid_dim_z_ + yi * voxel_grid_dim_z_ + zi];
            grid.val[4] = p_voxel_grid_TSDF_[xi * voxel_grid_dim_y_ * voxel_grid_dim_z_ + yi * voxel_grid_dim_z_ + (zi + 1)];
            grid.val[5] = p_voxel_grid_TSDF_[xi * voxel_grid_dim_y_ * voxel_grid_dim_z_ + (yi + 1) * voxel_grid_dim_z_ + (zi + 1)];
            grid.val[6] = p_voxel_grid_TSDF_[(xi + 1) * voxel_grid_dim_y_ * voxel_grid_dim_z_ + (yi + 1) * voxel_grid_dim_z_ + (zi + 1)];
            grid.val[7] = p_voxel_grid_TSDF_[(xi + 1) * voxel_grid_dim_y_ * voxel_grid_dim_z_ + yi * voxel_grid_dim_z_ + (zi + 1)];
            int cubeIndex = 0;
            if (grid.val[0] < 0) cubeIndex |= 1;
            if (grid.val[1] < 0) cubeIndex |= 2;
            if (grid.val[2] < 0) cubeIndex |= 4;
            if (grid.val[3] < 0) cubeIndex |= 8;
            if (grid.val[4] < 0) cubeIndex |= 16;
            if (grid.val[5] < 0) cubeIndex |= 32;
            if (grid.val[6] < 0) cubeIndex |= 64;
            if (grid.val[7] < 0) cubeIndex |= 128;
            XYZ vertlist[12];
            if (edgeTable[cubeIndex] == 0)
                continue;

            /* Find the vertices where the surface intersects the cube */
            if (edgeTable[cubeIndex] & 1)
                vertlist[0] =
                        VertexInterp(0, grid.p[0], grid.p[1], grid.val[0], grid.val[1]);
            if (edgeTable[cubeIndex] & 2)
                vertlist[1] =
                        VertexInterp(0, grid.p[1], grid.p[2], grid.val[1], grid.val[2]);
            if (edgeTable[cubeIndex] & 4)
                vertlist[2] =
                        VertexInterp(0, grid.p[2], grid.p[3], grid.val[2], grid.val[3]);
            if (edgeTable[cubeIndex] & 8)
                vertlist[3] =
                        VertexInterp(0, grid.p[3], grid.p[0], grid.val[3], grid.val[0]);
            if (edgeTable[cubeIndex] & 16)
                vertlist[4] =
                        VertexInterp(0, grid.p[4], grid.p[5], grid.val[4], grid.val[5]);
            if (edgeTable[cubeIndex] & 32)
                vertlist[5] =
                        VertexInterp(0, grid.p[5], grid.p[6], grid.val[5], grid.val[6]);
            if (edgeTable[cubeIndex] & 64)
                vertlist[6] =
                        VertexInterp(0, grid.p[6], grid.p[7], grid.val[6], grid.val[7]);
            if (edgeTable[cubeIndex] & 128)
                vertlist[7] =
                        VertexInterp(0, grid.p[7], grid.p[4], grid.val[7], grid.val[4]);
            if (edgeTable[cubeIndex] & 256)
                vertlist[8] =
                        VertexInterp(0, grid.p[0], grid.p[4], grid.val[0], grid.val[4]);
            if (edgeTable[cubeIndex] & 512)
                vertlist[9] =
                        VertexInterp(0, grid.p[1], grid.p[5], grid.val[1], grid.val[5]);
            if (edgeTable[cubeIndex] & 1024)
                vertlist[10] =
                        VertexInterp(0, grid.p[2], grid.p[6], grid.val[2], grid.val[6]);
            if (edgeTable[cubeIndex] & 2048)
                vertlist[11] =
                        VertexInterp(0, grid.p[3], grid.p[7], grid.val[3], grid.val[7]);

            /* Create the triangle */
            for (int ti = 0; triTable[cubeIndex][ti] != -1; ti += 3) {
                FACE f;
                TRIANGLE t;
                t.p[0] = vertlist[triTable[cubeIndex][ti]];
                t.p[1] = vertlist[triTable[cubeIndex][ti + 1]];
                t.p[2] = vertlist[triTable[cubeIndex][ti + 2]];
                for (int pi = 0; pi < 3; ++pi) {
                    std::string s = "x" + std::to_string(t.p[pi].x) + "y" + std::to_string(t.p[pi].y) + "z" + std::to_string(t.p[pi].z);
                    if (verticesIdx.find(s) == verticesIdx.end()) {
                        verticesIdx.insert(std::make_pair(s, vertexCount));
                        f.vIdx[pi] = vertexCount++;
                        t.p[pi].x = t.p[pi].x * voxel_size_ + voxel_grid_origin_x_;
                        t.p[pi].y = t.p[pi].y * voxel_size_ + voxel_grid_origin_y_;
                        t.p[pi].z = t.p[pi].z * voxel_size_ + voxel_grid_origin_z_;
                        vertices.push_back(t.p[pi]);
                    }
                    else
                        f.vIdx[pi] = verticesIdx[s];
                }
                faces.push_back(f);
            }
        }
        std::cout << vertexCount << std::endl;
        std::ofstream plyFile;
        plyFile.open(outputFileName);
        plyFile << "ply\nformat ascii 1.0\ncomment stanford bunny\nelement vertex ";
        plyFile << vertices.size() << "\n";
        plyFile << "property float x\nproperty float y\nproperty float z\n";// property uchar red\nproperty uchar green\nproperty uchar blue\n";
        plyFile << "element face " << faces.size() << "\n";
        plyFile << "property list int int vertex_index\nend_header\n";
        for (auto v : vertices) {
            plyFile << v.x << " " << v.y << " " << v.z << /*" " << (int)c.r << " " << (int)c.g << " " << (int)c.b <<*/ "\n";
        }
        for (auto f : faces) {
            plyFile << "3 " << f.vIdx[0] << " " << f.vIdx[1] << " " << f.vIdx[2] << "\n";
        }
        plyFile.close();
        std::cout << "File saved" << std::endl;
    }
}


